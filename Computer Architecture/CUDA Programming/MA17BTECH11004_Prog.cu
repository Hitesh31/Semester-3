#define N 512 
#define NUM_BLOCKS 16
#define NUM_THREADS 48
//Do not change above three lines.

//Submission should be named as  <RollNumber>_Prog.cu
//Upload just this cu file and nothing else. If you upload it as a zip, it will not be evaluated. 

/*Remember the following guidelines to avoid losing marks
This exercise is quite simple. The only tricky part is that total number of threads (NUM_BLOCKS*NUM_THREADS) may be different (higher or lower) from N.

Index of an array should not exceed the array size. 

No output array-element should be computed more than once

No marks will be given if the program does not compile or run (TAs will not debug your program at all)

Do not change the name of any variable that we have introduced. 
*/



#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void myadd(int *a,int *b,int *c)
{
	int k=((N*N)+NUM_BLOCKS*NUM_THREADS)/(NUM_BLOCKS*NUM_THREADS);
	int index=blockIdx.x*blockDim.x + threadIdx.x;
	for (int i=0;i<k;i++)
	{
		if((index*k+i)<N*N)
			 c[index*k+i]=a[index*k+i]+b[index*k+i];
	}
}

//TODO: WRITE GPU KERNEL. It should not be called repeatedly from the host, but just once. Each time it is called, it may process more than array-element or not process any array-element at all. 

int main (int argc, char **argv) {
  int A[N][N], B[N][N], C[N][N];
  int *d_A, *d_B, *d_C; // These are the copies of A, B and C on the GPU
  int *h_C;       // This is a host copy of the output of B from the GPU
  int i, j;
  int size = N*N*sizeof(int); //size of be allocated
  
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      A[i][j] = i+j;
      B[i][j]= 2*j-1;      
    }
  }

  // sequential implementation of main computation
  for(i=0;i<N;i++) {
    for(j=0;j<N;j++) {
      C[i][j] = A[i][j]+B[i][j];
    }
  }

  // TODO: ALLOCATE MEMORY FOR GPU COPIES OF d_A, d_B and d_C
  		hipMalloc((void **)&d_A,size);
  		hipMalloc((void **)&d_B,size);
  		hipMalloc((void **)&d_C,size);
  // TODO: COPY A TO d_A
  		hipMemcpy(d_A,A,size,hipMemcpyHostToDevice);
  // TODO: COPY B TO d_B
  		hipMemcpy(d_B,B,size,hipMemcpyHostToDevice);
  // TODO: CREATE BLOCKS with THREADS AND INVOKE GPU KERNEL
   //Use NUM_BLOCKS blocks, each with NUM_THREADS  threads
   		myadd <<<NUM_BLOCKS,NUM_THREADS>>>(d_A,d_B,d_C);
  // TODO: COPY d_C BACK FROM GPU to CPU in variable h_C
      h_C=(int *)malloc(size);
  		hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);
  // TODO: Verify result is correct by comparing
  //TODO: compare each element of h_C and C by subtracting them
   //print only those elements for which the above subtraction is non-zero
  int m=0;
  for(i=0;i<N;i++) 
  {
    for(j=0;j<N;j++) 
    {
    	if(h_C[N*i+j]-C[i][j]!=0)
    		{  
    			m++;
          		printf("%d \n",h_C[N*i+j]);
        	}
    }
  }
   if(m>0) printf("error\n");
    //IF even one element of h_C and C differ, report an error.
    //Otherwise, there is no error.
    //If your program is correct, no error should occur.
}

